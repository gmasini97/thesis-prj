#include "hip/hip_runtime.h"
#include "CmplxUtils.cuh"

__device__ __host__ hipComplex cuComplex_exp(float exp)
{
	float re = cos(exp);
	float im = sin(exp);
	return make_hipComplex(re, im);
}